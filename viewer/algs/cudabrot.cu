#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

constexpr int threads_per_block = 64;
constexpr int blocks_per_grid = 32;


__device__
inline int coord_to_px(const float co, const float min, const float max, const int size) {
    // Map coordinate to pixel.
    const float px = (co - min) / (max - min) * size;
    return (int)px;
}


__global__
void buddhabrot(int* img, const int width, const int height, const int iters, const int samples,
                           const float xmin, const float xmax, const float ymin, const float ymax,
                           hiprandState* states, int rand_seed) {
    // Initialize curand state.
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState* state = &states[tid];
    hiprand_init(rand_seed, tid, 0, state);

    for (int i = 0; i < samples; i++) {
        // Sample c value
        const float cx = (hiprand_uniform(state) - 0.5f) * 10.0f,
                    cy = (hiprand_uniform(state) - 0.5f) * 10.0f;

        // Iterate
        float zx = 0.0f, zy = 0.0f;
        bool in_set = true;
        for (int j = 0; j < iters; j++) {
            // Compute next z value
            float tmp = zx * zx - zy * zy + cx;
            zy = 2.0f * zx * zy + cy;
            zx = tmp;
            // Check divergence
            if (zx > 2.0f || zx < -2.0f ||
                zy > 2.0f || zy < -2.0f) {
                in_set = false;
                break;
            }
        }
        if (!in_set) {
            // Update values.
            float zx = 0.0f, zy = 0.0f;
            for (int j = 0; j <= iters; j++) {
                float tmp = zx * zx - zy * zy + cx;
                zy = 2.0f * zx * zy + cy;
                zx = tmp;
                const int px = coord_to_px(zx, xmin, xmax, width),
                          py = coord_to_px(zy, ymin, ymax, height);
                if (px >= 0 && px < width && py >= 0 && py < height) {
                    atomicAdd(&img[py * width + px], 1);
                }
                // Check divergence
                if (zx > 2.0f || zx < -2.0f ||
                    zy > 2.0f || zy < -2.0f) {
                    break;
                }
            }
        }
    }
}


// Usage: ./a.out
// Send to stdin: width height iters samples xmin xmax ymin ymax
// Image data will be written to stdout.
int main() {
    // Initialize curand.
    hiprandState* states;
    hipMalloc(&states, blocks_per_grid * threads_per_block * sizeof(hiprandState));
    int rand_seed = 0;

    while (true) {
        int width, height, iters, samples;
        float xmin, xmax, ymin, ymax;
        std::cin >> width >> height >> iters >> samples >> xmin >> xmax >> ymin >> ymax;

        // Allocate image.
        int* img = nullptr;
        hipMallocManaged(&img, width * height * sizeof(int));
        hipMemset(img, 0, width * height * sizeof(int));

        buddhabrot<<<blocks_per_grid, threads_per_block>>>(
            img, width, height, iters, samples, xmin, xmax, ymin, ymax, states, rand_seed
        );
        hipDeviceSynchronize();

        if (hipGetLastError() != hipSuccess) {
            std::cerr << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl;
            return 1;
        }

        // Write img to stdout.
        std::cout.write((char*)img, width * height * sizeof(int));
        std::cout.flush();

        hipFree(img);
        rand_seed += 1;
    }

    hipFree(states);
}
